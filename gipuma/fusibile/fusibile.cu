#include "hip/hip_runtime.h"
/* vim: ft=cpp
 * */

//#include <hip/hip_vector_types.h>
#ifdef _WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include "globalstate.h"
#include "algorithmparameters.h"
#include "camera.h"
#include "config.h"

#include <hip/hip_vector_types.h> // float4
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "vector_operations.h"
#include "point_cloud_list.h"

#define SAVE_TEXTURE
//#define SMOOTHNESS

#define FORCEINLINE __forceinline__

/* compute depth value from disparity or disparity value from depth
 * Input:  f         - focal length in pixel
 *         baseline  - baseline between cameras (in meters)
 *         d - either disparity or depth value
 * Output: either depth or disparity value
 */
__device__ FORCEINLINE float disparityDepthConversion_cu2 (
    const float &f, const Camera_cu &cam_ref, const Camera_cu &cam, const float &d )
{
    float baseline = l2_float4(cam_ref.C4 - cam.C4);
    return f * baseline / d;
}

__device__ FORCEINLINE void get3Dpoint_cu (
    const Camera_cu &cam, const int2 &p, const float &depth,
    float4 * __restrict__ ptX)
{
    // in case camera matrix is not normalized: see page 162, 
    // then depth might not be the real depth but w and depth needs to be computed from that first
    const float4 pt = make_float4 (
                                   depth * (float)p.x - cam.P_col34.x,
                                   depth * (float)p.y - cam.P_col34.y,
                                   depth - cam.P_col34.z,
                                   0);
    matvecmul4 (cam.M_inv, pt, ptX);
}

/* get angle between two vectors in 3D
 * Input: v1,v2 - vectors
 * Output: angle in radian
 */
__device__ FORCEINLINE float getAngle_cu ( const float4 &v1, const float4 &v2 ) {
    float angle = acosf ( dot4(v1, v2));
    return angle;
}

__device__ FORCEINLINE void project_on_camera (
    const float4 &X, const Camera_cu &cam, float2 *pt, float *depth)
{
    float4 tmp = make_float4 (0, 0, 0, 0);
    matvecmul4P (cam.P, X, (&tmp));
    pt->x = tmp.x / tmp.z;
    pt->y = tmp.y / tmp.z;

    *depth = tmp.z;
}

/*
 * Simple and fast depth math fusion based on depth map and normal consensus
 */
__global__ void fusibile (GlobalState &gs, int ref_camera)
{
    int2 p = make_int2 ( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y );

    const int cols = gs.cameras->cols;
    const int rows = gs.cameras->rows;

    if (p.x >= cols)
        return;
    if (p.y >= rows)
        return;

    const int center = p.y*cols+p.x;

    const CameraParameters_cu &camParams = *(gs.cameras);

    // if (gs.lines[ref_camera].used_pixels[center]==1)
    //     return;

    //printf("ref_camera is %d\n", ref_camera);
    const float4 normal = tex2D<float4> (gs.normal_depth_textures[ref_camera], p.x + 0.5f, p.y + 0.5f);
    //printf("Normal is %f %f %f\nDepth is %f\n", normal.x, normal.y, normal.z, normal.w);
    /*
     * For each point of the reference camera compute the 3d position corresponding to the corresponding depth.
     * Create a point only if the following conditions are fulfilled:
     * - Projected depths of other cameras does not differ more than gs.params.depthThresh
     * - Angle of normal does not differ more than gs.params.normalThresh
     */
    float depth = normal.w;

    float4 X;
    get3Dpoint_cu (camParams.cameras[ref_camera], p, depth, &X);
    float4 consistent_X = X;
    float4 consistent_normal = normal;
    float4 consistent_texture4 = tex2D<float4>(gs.color_images_textures[ref_camera], p.x+0.5f, p.y+0.5f);

    int number_consistent = 0;
    for ( int i = 0; i < camParams.viewSelectionSubsetNumber; i++ ) {

        int idxCurr = camParams.viewSelectionSubset[i];
        if (idxCurr == ref_camera)
            continue;

        // Project 3d point X on camera idxCurr
        float2 tmp_pt;
        project_on_camera (X, camParams.cameras[idxCurr], &tmp_pt, &depth);

        // Boundary check
        if (tmp_pt.x >=0 && tmp_pt.x < cols && tmp_pt.y >=0 && tmp_pt.y < rows) {
            // Compute interpolated depth and normal for tmp_pt w.r.t. camera ref_camera
            float4 tmp_normal_and_depth; // first 3 components normal, fourth depth
            tmp_normal_and_depth = tex2D<float4> (gs.normal_depth_textures[idxCurr], 
                tmp_pt.x+0.5f, tmp_pt.y+0.5f);

            const float depth_disp = disparityDepthConversion_cu2(
                camParams.cameras[ref_camera].K[0], 
                camParams.cameras[ref_camera], camParams.cameras[idxCurr],
                depth );
            
            const float tmp_depth_disp = disparityDepthConversion_cu2(
                camParams.cameras[ref_camera].K[0],
                camParams.cameras[ref_camera], camParams.cameras[idxCurr],
                tmp_normal_and_depth.w );
            
            // First consistency check on depth
            if (fabsf(depth_disp - tmp_depth_disp) < gs.params->depthThresh) {
                float angle = getAngle_cu (tmp_normal_and_depth, normal); // extract normal
                if (angle < gs.params->normalThresh) {
                    float4 tmp_X; // 3d point of consistent point on other view
                    int2 tmp_p = make_int2 ((int) tmp_pt.x, (int) tmp_pt.y);
                    get3Dpoint_cu (camParams.cameras[idxCurr], tmp_p, tmp_normal_and_depth.w, &tmp_X);

                    consistent_X = consistent_X + tmp_X;
                    consistent_normal = consistent_normal + tmp_normal_and_depth;
                    consistent_texture4 = consistent_texture4 
                        + tex2D<float4> (gs.color_images_textures[idxCurr], tmp_pt.x+0.5f, tmp_pt.y+0.5f);

                    number_consistent++;
                }
            }
        }
    }

    // Average normals and points
    consistent_X = consistent_X / ((float) number_consistent + 1.0f);
    consistent_normal = consistent_normal / ((float) number_consistent + 1.0f);
    consistent_texture4 = consistent_texture4 / ((float) number_consistent + 1.0f);

    if (number_consistent >= gs.params->numConsistentThresh) {
        gs.pc->points[center].coord  = consistent_X;
        gs.pc->points[center].normal = consistent_normal;
        gs.pc->points[center].texture4 = consistent_texture4;
    }
}

/* Copy point cloud to global memory */
void copy_point_cloud_to_host(GlobalState &gs, int cam, PointCloudList &pc_list)
{
    printf("Processing camera %d\n", cam);
    unsigned int count = pc_list.size;
    for (int y=0; y<gs.pc->rows; y++) {
        for (int x=0; x<gs.pc->cols; x++) {
            Point_cu &p = gs.pc->points[x+y*gs.pc->cols];
            const float4 X = p.coord;
            const float4 normal = p.normal;
            float texture4[4];
            texture4[0] = p.texture4.x;
            texture4[1] = p.texture4.y;
            texture4[2] = p.texture4.z;
            texture4[3] = p.texture4.w;

            if (count==pc_list.maximum) {
                printf("Not enough space to save points :'(\n... allocating more! :)");
                pc_list.increase_size(pc_list.maximum*2);
            }
            if (X.x != 0 && X.y != 0 && X.z != 0) {
                pc_list.points[count].coord = X;
                pc_list.points[count].normal = normal;
                pc_list.points[count].texture4[0] = texture4[0];
                pc_list.points[count].texture4[1] = texture4[1];
                pc_list.points[count].texture4[2] = texture4[2];
                pc_list.points[count].texture4[3] = texture4[3];
                count++;
            }
            p.coord = make_float4(0,0,0,0);
        }
    }
    printf("Found %.2f million points\n", count/1000000.0f);
    pc_list.size = count;
}

template< typename T >
void fusibile_cu(GlobalState &gs, PointCloudList &pc_list, int num_views)
{
    int rows = gs.cameras->rows;
    int cols = gs.cameras->cols;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("Run gipuma\n");

    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return ;
    }

    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }
    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA.\n");
        return ;
    }

    hipSetDevice(i);
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*128);

    dim3 grid_size_initrand;
    grid_size_initrand.x = (cols + 32-1)/32;
    grid_size_initrand.y = (rows + 32-1)/32;
    dim3 block_size_initrand;
    block_size_initrand.x = 32;
    block_size_initrand.y = 32;

    printf("Grid size initrand is grid: %d-%d block: %d-%d\n", 
        grid_size_initrand.x, grid_size_initrand.y, block_size_initrand.x, block_size_initrand.y);

    size_t avail;
    size_t total;
    hipMemGetInfo( &avail, &total );
    size_t used = total - avail;
    printf("Device memory used: %fMB\n", used/1000000.0f);

    //int shared_memory_size = sizeof(float)  * SHARED_SIZE ;
    printf("Fusing points\n");
    hipEventRecord(start);

    for (int cam=0; cam < num_views; cam++) {
        fusibile <<< grid_size_initrand, block_size_initrand, cam>>>(gs, cam);
        hipDeviceSynchronize();

        copy_point_cloud_to_host(gs, cam, pc_list); // slower but saves memory
        hipDeviceSynchronize();
    }

    hipEventRecord(stop);

    hipEventSynchronize(stop);

#if 1 // xxxx3333
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\t\tELAPSED %f seconds\n", milliseconds/1000.f);
#endif

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
}

int runcuda(GlobalState &gs, PointCloudList &pc_list, int num_views)
{
    printf("Run cuda\n");
    fusibile_cu<float4>(gs, pc_list, num_views);
    return 0;
}
