#include "hip/hip_runtime.h"
/* vim: ft=cpp
 * */

#include <stdio.h>
#include "globalstate.h"
#include "camera.h"
#include "config.h"

#include <hip/hip_vector_types.h>		// float4
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "point_cloud_list.h"
// #include <iostream>

#define FORCEINLINE __forceinline__

static __device__ float4 operator-(float4 a, float4 b)
{
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, 0);
}

static __device__ float4 operator+(float4 a, float4 b)
{
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, 0);
}

static __device__ float4 operator/(float4 a, float k)
{
	return make_float4(a.x / k, a.y / k, a.z / k, 0);
}

#define pow2(x) ((x)*(x))

static __device__ float l2_float4(float4 a)
{
	return sqrtf(pow2(a.x) + pow2(a.y) + pow2(a.z));

}

__device__ FORCEINLINE float depth_convert_cu(
	const float &f, // focal length
	const Camera_cu & cam_ref, 
	const Camera_cu & cam, const float &d)
{
	float baseline = l2_float4(cam_ref.C4 - cam.C4);
	return f * baseline / d;
}

#define matvecmul4(m, v, out) \
	out->x = m[0] * v.x + m[1] * v.y + m[2] * v.z; \
	out->y = m[3] * v.x + m[4] * v.y + m[5] * v.z; \
	out->z = m[6] * v.x + m[7] * v.y + m[8] * v.z;


__device__ FORCEINLINE void get_3dpoint_cu(
	const Camera_cu & cam,
	const int2 & p,
	const float &depth,
	float4 * __restrict__ ptX)
{
	// in case camera matrix is not normalized: see page 162, 
	// then depth might not be the real depth but w and depth needs to 
	// be computed from that first
	const float4 pt = make_float4(depth * (float) p.x - cam.P_col34.x,
								  depth * (float) p.y - cam.P_col34.y,
								  depth - cam.P_col34.z,
								  0);
	matvecmul4(cam.R_inv, pt, ptX);
}

#define matvecmul4P(m, v, out) \
out->x = m[0] * v.x + m[1] * v.y + m[2] * v.z + m[3]; \
out->y = m[4] * v.x + m[5] * v.y + m[6] * v.z + m[7]; \
out->z = m[8] * v.x + m[9] * v.y + m[10] * v.z + m[11];

__device__ FORCEINLINE void project_on_camera(const float4 & X, const Camera_cu & cam, float2 * pt, float *depth)
{
	float4 tmp = make_float4(0, 0, 0, 0);
	matvecmul4P(cam.P, X, (&tmp));
	pt->x = tmp.x / tmp.z;
	pt->y = tmp.y / tmp.z;

	*depth = tmp.z;
}

/*
 * Simple and fast depth math fusion based on depth map
 */
__global__ void fusibile(GlobalState & gs, int ref_camera)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, 
		blockIdx.y * blockDim.y + threadIdx.y);

	const int cols = gs.cameras->cols;
	const int rows = gs.cameras->rows;

	if (p.x >= cols)
		return;

	if (p.y >= rows)
		return;

	const int center = p.y * cols + p.x;
	const CameraParameters_cu & gs_cameras = *(gs.cameras);

	float4 sum_T = tex2D < float4 > (gs.color_images_textures[ref_camera],
											p.x + 0.5f, p.y + 0.5f);
	float depth = sum_T.w;
	if (depth <= 425.001) // 1.0/255.0 -- 0.0039
		return;

	float4 X;
	get_3dpoint_cu(gs_cameras.cameras[ref_camera], p, depth, &X);
	float4 sum_X = X;

	int count = 0;
	// gs.algorithm->consistent_threshold == 3
	int consistent_threshold = gs.algorithm->consistent_threshold;

	for (int i = 0; i < gs_cameras.n_cameras && count < 2*consistent_threshold; i++) {
		if (i == ref_camera)
			continue;

		// Project 3d point X on camera i
		float2 tmp_pt;
		project_on_camera(X, gs_cameras.cameras[i], &tmp_pt, &depth);

		// Boundary check
		if (tmp_pt.x < 0 || tmp_pt.x >= cols || tmp_pt.y < 0 || tmp_pt.y >= rows)
			continue;

		float4 tmp_T = tex2D < float4 > (gs.color_images_textures[i], 
			tmp_pt.x + 0.5f, tmp_pt.y + 0.5f);

		if (tmp_T.w <= 425.001) // 1.0/255.0 -- 0.0039
			continue;

		const float depth_disp = depth_convert_cu(
									gs_cameras.cameras[ref_camera].K[0], // focal_length
									gs_cameras.cameras[ref_camera],
									gs_cameras.cameras[i],
									depth);

		const float temp_disp = depth_convert_cu(
									gs_cameras.cameras[ref_camera].K[0],
									gs_cameras.cameras[ref_camera],
									gs_cameras.cameras[i],
									tmp_T.w);

		// check on depth
		if (fabsf(depth_disp - temp_disp) < gs.algorithm->depth_threshold) {
			// depth_threshold == 0.25
			float4 tmp_X;		// 3d point of consistent point on other view
			int2 tmp_p = make_int2((int) tmp_pt.x, (int) tmp_pt.y);
			get_3dpoint_cu(gs_cameras.cameras[i], tmp_p, tmp_T.w, &tmp_X);

			sum_X = sum_X + tmp_X;
			sum_T = sum_T + tmp_T;

			count++;
		}
	}

	if (count >= consistent_threshold) {
		// Average normals and points
		sum_X = sum_X/((float) count + 1.0f);
		sum_T = sum_T/((float) count + 1.0f);

		gs.pc->points[center].coord = sum_X;
		gs.pc->points[center].texture4 = sum_T;
	}
}

void dump_gpu_memory()
{
	size_t avail, total, used;
	hipMemGetInfo(&avail, &total);

	used = total - avail;
	printf("Device memory used: %.2f MB\n", used / 1000000.0f);
}

/* Copy point cloud to global memory */
void copy_pc_to_host(GlobalState & gs, int cam, PointCloudList & pc_list)
{
	printf("Processing camera %d\n", cam);

	int height = gs.cameras->rows;
	int width = gs.cameras->cols;
	unsigned int count = pc_list.size;

	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			Point_cu & p = gs.pc->points[x + y * width];

			if (count == pc_list.maximum) {
				pc_list.double_resize();
			}

			if (p.coord.x != 0 && p.coord.y != 0 && p.coord.z != 0) {
				pc_list.points[count].coord = p.coord;
				pc_list.points[count].texture4[0] = p.texture4.x;
				pc_list.points[count].texture4[1] = p.texture4.y;
				pc_list.points[count].texture4[2] = p.texture4.z;
				pc_list.points[count].texture4[3] = p.texture4.w;
				count++;
			}
		}
	}
	pc_list.size = count;

	printf("Found %.2fM points\n", count / 1000000.0f);
}

void fusibile_cu(GlobalState & gs, PointCloudList & pc_list, int num_views)
{
	int rows = gs.cameras->rows;
	int cols = gs.cameras->cols;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	printf("Run gipuma\n");

	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return;
	}

	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return;
	}

	hipSetDevice(i);
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024 * 128);

	dim3 grid_size;
	grid_size.x = (cols + 32 - 1) / 32;
	grid_size.y = (rows + 32 - 1) / 32;

	dim3 block_size;
	block_size.x = 32;
	block_size.y = 32;

	printf("Grid size: %d-%d block: %d-%d\n", grid_size.x, grid_size.y, block_size.x, block_size.y);

	dump_gpu_memory();

	//int shared_memory_size = sizeof(float)  * SHARED_SIZE ;
	printf("Fusing points\n");
	hipEventRecord(start);

	for (int cam = 0; cam < num_views; cam++) {
		fusibile <<< grid_size, block_size, cam >>> (gs, cam);
		hipDeviceSynchronize();

		copy_pc_to_host(gs, cam, pc_list);	// slower but saves memory
		hipDeviceSynchronize();
	}

	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\tELAPSED %f seconds\n", milliseconds / 1000.f);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
}

int run_cuda(GlobalState & gs, PointCloudList & pc_list, int num_views)
{
	printf("Run cuda\n");
	fusibile_cu(gs, pc_list, num_views);

	return 0;
}
